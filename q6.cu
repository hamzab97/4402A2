#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cstdio>
#include <chrono>

__global__ void minplus(int n, int* x, int *y)
//multiply a and b, store result in c, copy result back to a after
//min plus is c(i, j) = min from k = 1 to k = n (a(i,k) + b(k,j))
{
  // printf("N= %d\n", N);
  int j = threadIdx.y + (blockIdx.y * blockDim.y); //get the row
	int i= threadIdx.x + (blockIdx.x * blockDim.x); //get the col
	if (i < n and j < n){
		for (int k = 0; k < n; k++){
			x[i*n + j] = min(x[i*n + j], y[i*n + k] + y[k * n + j]);
      //printf("i: %d. j: %d. k: %d. value is: %d\n", i, j, k, x[i*n + j]);
		}

	}
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

void fw(int n, int* path){
  for (int k = 0; k < n; k++){
		for (int i = 0; i < n; i++){
			for (int j = 0; j < n; j++){
				path[i*n + j] = min ( path[i*n + j], path[i*n + k]+path[k*n + j] );
			}
		}
	}
}

int main(void)
{
  std::cout << "started " << '\n';
  int N = 16;
  int *a, *b, *d_a, *d_b;
  a = (int*)malloc(N*sizeof(int));
  b = (int*)malloc(N*sizeof(int));
  hipMalloc((void**)&d_a, N*sizeof(int));
  hipMalloc((void**)&d_b, N*sizeof(int));

  checkErrors("memory allocation");

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++){
      int ran = rand()%(30-0 + 1) + 0;
      // std::cout << "ran is %d"<< ran << '\n';
      a[i*N + j] = ran;
      b[i*N + j] = ran;
      //std::cout << "i: " << i << " j: " <<j<< "value is " << a[i*N + j] << '\n';
    }

  }
  std::cout << "a before cuda" << '\n';
  // int maxError = 0.0f;
  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      //std::cout << a[i*N + j] << '\n';
    }
  }

  hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  checkErrors("copy data to device");

  std::cout << "called cuda" << '\n';

  // number of threads per block
  int numThreadsPerBlock = 16;
  int numBlocks = (N+numThreadsPerBlock-1) / numThreadsPerBlock;

  //setup kernal launch parameters
  dim3 THREADS(numThreadsPerBlock, numThreadsPerBlock);
  dim3 BLOCKS(numBlocks, numBlocks);

  // minplus<<<BLOCKS, THREADS>>>(N, d_a, d_b);
	auto cuda_t1 = std::chrono::high_resolution_clock::now(); //start timer
	for (int i = 0; i < N; i++){
    // Perform minplus
		  minplus<<<numBlocks, numThreadsPerBlock>>>(N, d_a, d_b);
      checkErrors("compute on device");
      hipMemcpy(a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);
      checkErrors("copy data from device");

			hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
			checkErrors("copy data to device");
	}
	auto cuda_t2 = std::chrono::high_resolution_clock::now(); //end timer

  int *h_z = (int*)malloc(N*sizeof(int));
  hipMemcpy(h_z, d_a, N*sizeof(int), hipMemcpyDeviceToHost);
  checkErrors("copy data from device");

  std::cout << "cuda finished" << '\n';

  // int maxError = 0.0f;
  // for (int i = 0; i < N; i++){

  // }
    // maxError = max(maxError, abs(y[i]-4.0f));

		// printf(a[i]);
	// printf("Max error: %f\n", maxError);

  // std::cout << "done printing" << '\n';


	std::cout << "calling serial FW" << '\n';
	auto serial_t1 = std::chrono::high_resolution_clock::now(); //end timer
	fw(N, b);
	auto serial_t2 = std::chrono::high_resolution_clock::now(); //end timer
	std::cout << "serial finished" << '\n';

	auto duration_cuda = std::chrono::duration_cast<std::chrono::microseconds>( cuda_t2 - cuda_t1 ).count();
	auto duration_serial = std::chrono::duration_cast<std::chrono::microseconds>( serial_t2 - serial_t1 ).count();

	std::cout << "cuda duration: "<<duration_cuda << " serial duation: " <<duration_serial << '\n';

	for (int j = 0; j < N; j++){
		std::cout << "i: " << i << " j: " <<j<< "value from cuda is " << h_z[i*N + j] << " value from serial is " <<b[i*N + j]<< '\n';
	}

  hipFree(d_a);
  hipFree(d_b);
  free(h_z);
  free(a);
  free(b);

  std::cout << "fnished" << '\n';
}
