#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void minplus(int N, int* a, int *b)
//multiply a and b, store result in c, copy result back to a after
//min plus is c(i, j) = min from k = 1 to k = n (a(i,k) + b(k,j))
{
  int j = threadIdx.y + (blockId.y * blockDim.y); //get the row
	int i= threadIdx.x + (blockId.x * blockDim.x); //get the col
	if (i < N and j < N){
		for (int k = 0; k < N; k++){
			a[i][j] = min(a[i][j], b[i][k] + b[k][j]);
		}
	}
}

int main(void)
{
  int N = 32;
  int *a, *b, *d_a, *d_b;
  a = (int*)malloc(N*sizeof(int));
  b = (int*)malloc(N*sizeof(int));
  hipMalloc(&d_a, N*sizeof(int));
  hipMalloc(&d_b, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    a[i] = rand();
    b[i] = a[i];
  }

  hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  // Perform minplus
	for (int i = 0; i < N; i++){
		  minplus<<<(N+255)/256, 256>>>(N, d_a, d_b);
	}


  hipMemcpy(a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);

  // int maxError = 0.0f;
  for (int i = 0; i < N; i++)
    // maxError = max(maxError, abs(y[i]-4.0f));
		printf(a[i]);
	printf("Max error: %f\n", maxError);


  hipFree(d_a);
  hipFree(d_b);
  free(a);
  free(b);
}
