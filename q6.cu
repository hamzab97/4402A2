#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdio>


__global__ void minplus(int N, int* a, int *b)
//multiply a and b, store result in c, copy result back to a after
//min plus is c(i, j) = min from k = 1 to k = n (a(i,k) + b(k,j))
{
  printf("N= %d\n", N);
  int j = threadIdx.y + (blockIdx.y * blockDim.y); //get the row
	int i= threadIdx.x + (blockIdx.x * blockDim.x); //get the col
	if (i < N and j < N){
		for (int k = 0; k < N; k++){
			a[i*N + j] = min(a[i*N + j], b[i*N + k] + b[k * N + j]);
		}
	}
}

int main(void)
{
  int N = 4;
  int *a, *b, *d_a, *d_b;
  a = (int*)malloc(N*sizeof(int));
  b = (int*)malloc(N*sizeof(int));
  hipMalloc(&d_a, N*sizeof(int));
  hipMalloc(&d_b, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    a[i] = rand();
    b[i] = a[i];
  }

  hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  // Perform minplus
	for (int i = 0; i < N; i++){
		  minplus<<<(N+255)/256, 256>>>(N, d_a, d_b);
	}


  hipMemcpy(a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);

  // int maxError = 0.0f;
  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      std::cout << a[(i * N) + j] << '\n'[j==N-1];
    }
  }
    // maxError = max(maxError, abs(y[i]-4.0f));

		// printf(a[i]);
	// printf("Max error: %f\n", maxError);


  hipFree(d_a);
  hipFree(d_b);
  free(a);
  free(b);
}
