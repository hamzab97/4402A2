#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cstdio>

// number of threads per block
static int numThreadsPerBlock = 256;

__global__ void minplus(int n, int* x, int *y)
//multiply a and b, store result in c, copy result back to a after
//min plus is c(i, j) = min from k = 1 to k = n (a(i,k) + b(k,j))
{
  // printf("N= %d\n", N);
  int j = threadIdx.y + (blockIdx.y * blockDim.y); //get the row
	int i= threadIdx.x + (blockIdx.x * blockDim.x); //get the col
	if (i < n and j < n){
		for (int k = 0; k < n; k++){
			x[i*n + j] = min(x[i*n + j], y[i*n + k] + y[k * n + j]);
		}
    std::cout << "i: " << i << " j: " << "value is " << x[i*n + j] << '\n';
	}
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}


int main(void)
{
  std::cout << "started " << '\n';
  int N = 4;
  int *a, *b, *d_a, *d_b;
  a = (int*)malloc(N*sizeof(int));
  b = (int*)malloc(N*sizeof(int));
  hipMalloc((void**)&d_a, N*sizeof(int));
  hipMalloc((void**)&d_b, N*sizeof(int));

  checkErrors("memory allocation");

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++){
      int ran = rand()%(30-0 + 1) + 0;
      // std::cout << "ran is %d"<< ran << '\n';
      a[i*N + j] = ran;
      b[i*N + j] = ran;
      std::cout << "i: " << i << " j: " << "value is " << a[i*n + j] << '\n';
    }

  }
  std::cout << "a before cuda" << '\n';
  // int maxError = 0.0f;
  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      std::cout << a[i*N + j] << '\n';
    }
  }

  hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  checkErrors("copy data to device");

  std::cout << "called cuda" << '\n';

  // Perform minplus
  int numBlocks = (N+numThreadsPerBlock-1) / numThreadsPerBlock;
	for (int i = 0; i < N; i++){
		  minplus<<<numBlocks, numThreadsPerBlock>>>(N, d_a, d_b);
      checkErrors("compute on device");
	}

  int *h_z = (int*)malloc(N*sizeof(int));
  hipMemcpy(h_z, d_a, N*sizeof(int), hipMemcpyDeviceToHost);
  checkErrors("copy data from device");

  std::cout << "a after cuda" << '\n';

  // int maxError = 0.0f;
  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      std::cout << "i: " << i << " j: " << "value is " << h_z[i*n + j] << '\n';
    }
  }
    // maxError = max(maxError, abs(y[i]-4.0f));

		// printf(a[i]);
	// printf("Max error: %f\n", maxError);

  std::cout << "done printing" << '\n';
  hipFree(d_a);
  hipFree(d_b);
  free(h_z);
  free(a);
  free(b);

  std::cout << "fnished" << '\n';
}
