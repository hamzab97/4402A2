
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <cstdio>


__global__ void minplus(int N, int* a, int *b)
//multiply a and b, store result in c, copy result back to a after
//min plus is c(i, j) = min from k = 1 to k = n (a(i,k) + b(k,j))
{
  // printf("N= %d\n", N);
  int j = threadIdx.y + (blockIdx.y * blockDim.y); //get the row
	int i= threadIdx.x + (blockIdx.x * blockDim.x); //get the col
	if (i < N and j < N){
		for (int k = 0; k < N; k++){
			a[i*N + j] = min(a[i*N + j], b[i*N + k] + b[k * N + j]);
		}
	}
}

int main(void)
{
  std::cout << "started " << '\n';
  int N = 4;
  int *a, *b, *d_a, *d_b;
  a = (int*)malloc(N*sizeof(int));
  b = (int*)malloc(N*sizeof(int));
  hipMalloc(&d_a, N*sizeof(int));
  hipMalloc(&d_b, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    int ran = rand();
    a[i] = ran;
    b[i] = ran;
  }
  std::cout << "a before cuda" << '\n';
  // int maxError = 0.0f;
  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      std::cout << a[i*N + j] << '\n';
    }
  }

  hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  std::cout << "called cuda" << '\n';

  // Perform minplus
	for (int i = 0; i < N; i++){
		  minplus<<<(N+255)/256, 256>>>(N, d_a, d_b);
	}


  hipMemcpy(a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "a after cuda" << '\n';

  // int maxError = 0.0f;
  for (int i = 0; i < N; i++){
    for (int j = 0; j < N; j++){
      std::cout << a[i*N + j] << '\n';
    }
  }
    // maxError = max(maxError, abs(y[i]-4.0f));

		// printf(a[i]);
	// printf("Max error: %f\n", maxError);


  hipFree(d_a);
  hipFree(d_b);
  free(a);
  free(b);

  std::cout << "fnished" << '\n';
}
