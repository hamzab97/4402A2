
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <cstdio>

// number of threads per block
static int numThreadsPerBlock = 256;

__global__ void minplus(int n, int* x, int *y)
//multiply a and b, store result in c, copy result back to a after
//min plus is c(i, j) = min from k = 1 to k = n (a(i,k) + b(k,j))
{
  // printf("N= %d\n", N);
  int j = threadIdx.y + (blockIdx.y * blockDim.y); //get the row
	int i= threadIdx.x + (blockIdx.x * blockDim.x); //get the col
	if (i < n and j < n){
		for (int k = 0; k < n; k++){
			x[i*n + j] = min(x[i*n + j], y[i*n + k] + y[k * n + j]);
		}
	}
}

int main(void)
{
  std::cout << "started " << '\n';
  int N = 6;
  int *a, *b, *d_a, *d_b;
  a = (int*)malloc(N*sizeof(int));
  b = (int*)malloc(N*sizeof(int));
  hipMalloc((void**)&d_a, N*sizeof(int));
  hipMalloc((void**)&d_b, N*sizeof(int));

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++){
      int ran = rand()%(10-0 + 1) + 0;
      a[i*N + j] = ran;
      b[i*N + j] = ran;
    }

  }
  // std::cout << "a before cuda" << '\n';
  // // int maxError = 0.0f;
  // for (int i = 0; i < N; i++){
  //   for (int j = 0; j < N; j++){
  //     std::cout << a[i*N + j] << '\n';
  //   }
  // }

  hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  std::cout << "called cuda" << '\n';

  // Perform minplus
  int numBlocks = (N+numThreadsPerBlock-1) / numThreadsPerBlock;
	for (int i = 0; i < N; i++){
		  minplus<<<numBlocks, numThreadsPerBlock>>>(N, d_a, d_b);
	}

  int *h_z = (int*)malloc(N*sizeof(int));
  hipMemcpy(h_z, d_a, N*sizeof(int), hipMemcpyDeviceToHost);

  // std::cout << "a after cuda" << '\n';
  //
  // // int maxError = 0.0f;
  // for (int i = 0; i < N; i++){
  //   for (int j = 0; j < N; j++){
  //     std::cout << a[i*N + j] << '\n';
  //   }
  // }
    // maxError = max(maxError, abs(y[i]-4.0f));

		// printf(a[i]);
	// printf("Max error: %f\n", maxError);


  hipFree(d_a);
  hipFree(d_b);
  free(h_z);
  free(a);
  free(b);

  std::cout << "fnished" << '\n';
}
